#include "hip/hip_runtime.h"
/* 
 * Copyright (C) 2012 Chris McClelland
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *  
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
#include "mandel.h"

// Inner loop of all the renderers. Accepts a point x0+iy0 on the complex plane
// and returns an eight-bit color index for the corresponding pixel. This is the
// critical section from a performance point of view, because the loop can
// execute up to 256 times, and this must be done for each and every pixel.
//
// The compiler generates two versions of this function, one on the host side
// and another functionally identical one on the GPU side.
//
__host__
__device__
unsigned char getPixColor(const float x0, const float y0) {
	float x = 0.0f, y = 0.0f, tmpx;
	int i = 0;
	while ( x*x + y*y < 4.0f && i < 256 ) {
		tmpx = x*x - y*y + x0;
		y = 2*x*y + y0;
		x = tmpx;
		i++;
	}
	return (unsigned char)i;
}

// A naive CPU-based renderer, which just recalculates the point on the complex
// plane and the address in screen memory for each and every pixel.
//
void cpuNaiveRender(
	unsigned char *pixels, const float xCentre, const float yCentre,
	const float size)
{
	const float delta = size/SCREEN_WIDTH;
	int px, py;
	float x0, y0;
	for ( py = 0; py < SCREEN_HEIGHT; py++ ) {
		for ( px = 0; px < SCREEN_WIDTH; px++ ) {
			x0 = xCentre + delta*(px - SCREEN_WIDTH/2);
			y0 = yCentre + delta*(py - SCREEN_HEIGHT/2);
			pixels[SCREEN_WIDTH*py + px] = getPixColor(x0, y0);
		}
	}
}

// A more efficient CPU-based renderer, which avoids recalculation by just
// adding the known delta each time. Unfortunately, because getPixColor() is
// where the CPU spends most of its time, in practice this optimization is not
// particularly great.
//
void cpuOptRender(
	unsigned char *pixels, const float xCentre, const float yCentre,
	const float size)
{
	const float delta = size/SCREEN_WIDTH;
	int px, py;
	float x0, y0;
	y0 = yCentre - SCREEN_HEIGHT*delta/2;
	for ( py = 0; py < SCREEN_HEIGHT; py++ ) {
		x0 = xCentre - SCREEN_WIDTH*delta/2;
		for ( px = 0; px < SCREEN_WIDTH; px++ ) {
			*pixels++ = getPixColor(x0, y0);
			x0 += delta;
		}
		y0 += delta;
	}
}

// On the way to developing a GPU renderer, it's first necessary to think about
// how to partition the GPU resources into blocks of threads. Because it's
// easier to get this right on a CPU than a GPU, it makes sense to implement
// the partitioned algorithm on the CPU first.
//
// In this case, the screen is split up into tiles of 32 pixels by 16. Each tile
// can then be worked on by a GPU thread-block (a block can contain up to 512
// threads).
//
void cpuTiledRender(
	unsigned char *pixels, const float xCentre, const float yCentre,
	const float size)
{
	const float delta = size/SCREEN_WIDTH;
	int tx, ty, px, py;
	int xPix, yPix;
	float x0, y0;
	for ( ty = 0; ty < VERT_TILES; ty++ ) {
		for ( tx = 0; tx < HORIZ_TILES; tx++ ) {
			for ( py = 0; py < TILE_HEIGHT; py++ ) {
				for ( px = 0; px < TILE_WIDTH; px++ ) {
					xPix = tx*TILE_WIDTH + px;
					yPix = ty*TILE_HEIGHT + py;
					x0 = xCentre + delta*(xPix - SCREEN_WIDTH/2);
					y0 = yCentre + delta*(yPix - SCREEN_HEIGHT/2);
					pixels[SCREEN_WIDTH*yPix + xPix] = getPixColor(x0, y0);
				}
			}
		}
	}
}

// A GPU renderer. Note the similarity to the code in the inner loop of the
// CPU tiled renderer.
//
__global__
void mandelKernel(
	const float xCentre,     // the real component of the region to render
	const float yCentre,     // the imaginary component of the region to render
	const float delta,       // the amount to step for each pixel
	unsigned char *gpuArray  // an array to store the results
) {
	const int xPix = blockDim.x * blockIdx.x + threadIdx.x;
	const int yPix = blockDim.y * blockIdx.y + threadIdx.y;
	const float x0 = xCentre + delta*(xPix - SCREEN_WIDTH/2);
	const float y0 = yCentre + delta*(yPix - SCREEN_HEIGHT/2);
	gpuArray[SCREEN_WIDTH*yPix + xPix] = getPixColor(x0, y0);
}

void gpuRender(
	unsigned char *pixels, const float xCentre, const float yCentre,
	const float size)
{
	const float delta = size/SCREEN_WIDTH;
	const dim3 blocksPerGrid(HORIZ_TILES, VERT_TILES);
	const dim3 threadsPerBlock(TILE_WIDTH, TILE_HEIGHT);
	mandelKernel<<<blocksPerGrid, threadsPerBlock>>>(xCentre, yCentre, delta, pixels);
}	

unsigned char *gpuAlloc(void) {
	unsigned char *gpuArray;
	hipError_t err = hipMalloc(&gpuArray, ARRAY_SIZE);
	if ( err != hipSuccess ) return NULL;
	return gpuArray;
}

int gpuBlit(const unsigned char *sourceArray, unsigned char *destArray) {
	hipError_t err =
		hipMemcpy(destArray, sourceArray, ARRAY_SIZE, hipMemcpyDeviceToHost);
	if ( err != hipSuccess ) return 1;
	return 0;
}

void gpuFree(unsigned char *gpuArray) {
	hipFree(gpuArray);
}
